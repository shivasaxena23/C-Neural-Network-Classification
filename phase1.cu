
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include <math.h>
#include<sys/time.h>

__global__ void MatrixTranspose(float *a,float *b,int nx, int ny){
int ix = threadIdx.x+ blockIdx.x*blockDim.x;
int iy = threadIdx.y+ blockIdx.y*blockDim.y;
int idx = ix*ny + iy;
int odx= iy*nx + ix;

if((ix<nx)&&(iy<ny)){
			b[odx]=a[idx];
	}

}


__global__ void MatrixMul(float *a,float *b, float *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

__global__ void MatAdd(float *A, float *B, float *C, int nx, int ny){
int ix = threadIdx.x+ blockIdx.x*blockDim.x;
int iy = threadIdx.y+ blockIdx.y*blockDim.y;
int idx = ix*ny + iy;

if((ix<nx)&&(iy<ny)){
			C[idx]=A[idx]+B[idx];
	}

}

__global__ void Mul(float *A, float *B, float *C, int nx, int ny){
int ix = threadIdx.x+ blockIdx.x*blockDim.x;
int iy = threadIdx.y+ blockIdx.y*blockDim.y;
int idx = ix*ny + iy;

if((ix<nx)&&(iy<ny)){
			C[idx]=A[idx]*B[idx];
	}
}

__global__ void div(float *A, float *B, float *C, int nx, int ny){
int ix = threadIdx.x+ blockIdx.x*blockDim.x;
int iy = threadIdx.y+ blockIdx.y*blockDim.y;
int idx = ix*ny + iy;

if((ix<nx)&&(iy<ny)){
			C[idx]=A[idx]/B[idx];
	}
}

__global__ void MatSub(float *A, float *B, float *C, int nx, int ny){
int ix = threadIdx.x+ blockIdx.x*blockDim.x;
int iy = threadIdx.y+ blockIdx.y*blockDim.y;
int idx = ix*ny + iy;

if((ix<nx)&&(iy<ny)){
			C[idx]=A[idx]-B[idx];
	}
}

double getTimeStamp(){
struct timeval tv;
gettimeofday(&tv, NULL);
return (double) tv.tv_usec/1000000+ tv.tv_sec;
}

struct matStruct{
float *m;
int x;
int y;
};
typedef struct matStruct matrix;

matrix setup_matrix(int x,int y)
{
matrix p;
p.m= (float *)malloc(x*y*sizeof(float *));
p.x=x;
p.y=y;

return p;
}
matrix transpose(matrix A){
	matrix C;
	C=setup_matrix(A.y,A.x);
	float *d_A, *d_C;
	hipMalloc((void **) &d_A, ((A.x*A.y)*sizeof(float)));
	hipMalloc((void **) &d_C, ((C.x*C.y)*sizeof(float)));
	hipMemcpy(d_A,A.m, (A.x*A.y)*sizeof(float), hipMemcpyHostToDevice );
	dim3 block(32,32);
	dim3 grid(1,1);
	MatrixTranspose<<<grid,block>>>(d_A,d_C,A.x,A.y);
	hipMemcpy(C.m,d_C,(C.x*C.y)*sizeof(float), hipMemcpyDeviceToHost);
	return C;
}



matrix matmul(matrix A, matrix B){
	if ((A.y==B.x))
	{	matrix C;
		C=setup_matrix(A.x,B.y);
		float *d_A, *d_B, *d_C;
		
				
		hipMalloc((void **) &d_A, ((A.x*A.y)*sizeof(float)));
		hipMalloc((void **) &d_B, ((B.x*B.y)*sizeof(float)));
		hipMalloc((void **) &d_C, ((C.x*C.y)*sizeof(float)));
		hipMemcpy(d_A,A.m, (A.x*A.y)*sizeof(float), hipMemcpyHostToDevice );
		hipMemcpy(d_B,B.m, (B.x*B.y)*sizeof(float), hipMemcpyHostToDevice );
		printf("%d %d\n",C.x,C.y);
		dim3 block(32,32);
		dim3 grid(1,1);
		MatrixMul<<<grid,block>>>(d_A,d_B,d_C,A.x,A.y,B.y);
		
		hipMemcpy(C.m,d_C,(C.x*C.y)*sizeof(float), hipMemcpyDeviceToHost);
		return C;

		
	}
	else{
	printf("Error:Vector Sum failed incompatible sizes");
	matrix C;
	C=setup_matrix(A.x,A.y);
	return C;
	}
	
}


matrix add_mat(matrix A, matrix B){
	if ((A.x==B.x)&&(A.y==B.y))
	{	matrix C;
		C=setup_matrix(A.x,A.y);
		float *d_A, *d_B, *d_C;
		float *h_dC;
		int bytes=(A.x*A.y)*sizeof(float);
		hipError_t status3 = hipHostMalloc((void**)&h_dC, bytes, hipHostMallocDefault);
		
		hipMalloc((void **) &d_A, bytes);
		hipMalloc((void **) &d_B, bytes);
		hipMalloc((void **) &d_C,bytes);
		hipMemcpy(d_A,A.m, bytes, hipMemcpyHostToDevice );
		hipMemcpy(d_B,B.m, bytes, hipMemcpyHostToDevice );
		
		dim3 block(A.x,A.y);
		dim3 grid(1,1);
		MatAdd<<<grid,block>>>(d_A,d_B,d_C, A.x,A.y);
		hipDeviceSynchronize();
		hipMemcpy(C.m,d_C,bytes, hipMemcpyDeviceToHost);
		
		return C;

		
	}
	else{
	printf("Error:Vector Sum failed incompatible sizes");
	matrix C;
	C=setup_matrix(A.x,A.y);
	return C;
	}
	
}

matrix multiply(matrix A, matrix B){
	if ((A.x==B.x)&&(A.y==B.y))
	{	matrix C;
		C=setup_matrix(A.x,A.y);
		float *d_A, *d_B, *d_C;
		float *h_dC;
		int bytes=(A.x*A.y)*sizeof(float);
		hipError_t status3 = hipHostMalloc((void**)&h_dC, bytes, hipHostMallocDefault);
		
		hipMalloc((void **) &d_A, bytes);
		hipMalloc((void **) &d_B, bytes);
		hipMalloc((void **) &d_C,bytes);
		hipMemcpy(d_A,A.m, bytes, hipMemcpyHostToDevice );
		hipMemcpy(d_B,B.m, bytes, hipMemcpyHostToDevice );
		
		dim3 block(A.x,A.y);
		dim3 grid(1,1);
		Mul<<<grid,block>>>(d_A,d_B,d_C, A.x,A.y);
		hipDeviceSynchronize();
		hipMemcpy(C.m,d_C,bytes, hipMemcpyDeviceToHost);
		return C;

		
	}
	else{
	printf("Error:Vector Sum failed incompatible sizes");
	matrix C;
	C=setup_matrix(A.x,A.y);
	return C;
	}
	
}

matrix sub_mat(matrix A, matrix B){
	if ((A.x==B.x)&&(A.y==B.y))
	{	matrix C;
		C=setup_matrix(A.x,A.y);
		float *d_A, *d_B, *d_C;
		float *h_dC;
		int bytes=(A.x*A.y)*sizeof(float);
		hipError_t status3 = hipHostMalloc((void**)&h_dC, bytes, hipHostMallocDefault);
		
		hipMalloc((void **) &d_A, bytes);
		hipMalloc((void **) &d_B, bytes);
		hipMalloc((void **) &d_C,bytes);
		hipMemcpy(d_A,A.m, bytes, hipMemcpyHostToDevice );
		hipMemcpy(d_B,B.m, bytes, hipMemcpyHostToDevice );
		
		dim3 block(A.x,A.y);
		dim3 grid(1,1);
		MatSub<<<grid,block>>>(d_A,d_B,d_C, A.x,A.y);
		hipDeviceSynchronize();
		hipMemcpy(C.m,d_C,bytes, hipMemcpyDeviceToHost);
		return C;

		
	}
	else{
	printf("Error:Vector Sum failed incompatible sizes");
	matrix C;
	C=setup_matrix(A.x,A.y);
	return C;
	}
	
}

matrix divide(matrix A, matrix B){
	if ((A.x==B.x)&&(A.y==B.y))
	{	matrix C;
		C=setup_matrix(A.x,A.y);
		float *d_A, *d_B, *d_C;
		float *h_dC;
		int bytes=(A.x*A.y)*sizeof(float);
		hipError_t status3 = hipHostMalloc((void**)&h_dC, bytes, hipHostMallocDefault);
		
		hipMalloc((void **) &d_A, bytes);
		hipMalloc((void **) &d_B, bytes);
		hipMalloc((void **) &d_C,bytes);
		hipMemcpy(d_A,A.m, bytes, hipMemcpyHostToDevice );
		hipMemcpy(d_B,B.m, bytes, hipMemcpyHostToDevice );
		
		dim3 block(A.x,A.y);
		dim3 grid(1,1);
		div<<<grid,block>>>(d_A,d_B,d_C, A.x,A.y);
		hipDeviceSynchronize();
		hipMemcpy(C.m,d_C,bytes, hipMemcpyDeviceToHost);
		return C;

		
	}
	else{
	printf("Error:Vector Sum failed incompatible sizes");
	matrix C;
	C=setup_matrix(A.x,A.y);
	return C;
	}
	
}


int main( int argc, char *argv[])
{
if (argc !=3){
	printf("Error: wrong number of args\n");
	exit(0);
}
int nx= atoi(argv[1]);
int ny = atoi( argv[2]);
matrix a1,b1,c1;
a1=setup_matrix(3,3);
b1=setup_matrix(3,6);

for(int i=0; i<a1.x; i++)
{for(int j=0;j<a1.y;j++)
{a1.m[a1.y*i+j]=2;
printf("%f ",a1.m[a1.y*i+j]);
}
printf("\n");
}
for(int i=0; i<b1.x; i++)
{for(int j=0;j<b1.y;j++)
{b1.m[b1.y*i+j]=1;
printf("%f ",b1.m[b1.y*i+j]);
}
printf("\n");
}
c1=matmul(a1,b1);

for(int i=0; i<c1.x; i++)
{for(int j=0;j<c1.y;j++)
{printf("%f ",c1.m[c1.y*i+j]);
}
printf("\n");
}

}
